
#include <hip/hip_runtime.h>
__global__ void test_shf_l_clamp(unsigned* a, unsigned* b, unsigned* c, unsigned* d){
    asm("shf.l.clamp.b32 %0, %1, %2, %3;" : "=r"(*d) : "r"(*a), "r"(*b), "r"(*c));
}

__global__ void test_shf_r_clamp(unsigned* a, unsigned* b, unsigned* c, unsigned* d){
    asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(*d) : "r"(*a), "r"(*b), "r"(*c));
}

__global__ void test_shf_l_wrap(unsigned* a, unsigned* b, unsigned* c, unsigned* d){
    asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(*d) : "r"(*a), "r"(*b), "r"(*c));
}

__global__ void test_shf_r_wrap(unsigned* a, unsigned* b, unsigned* c, unsigned* d){
    asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(*d) : "r"(*a), "r"(*b), "r"(*c));
}