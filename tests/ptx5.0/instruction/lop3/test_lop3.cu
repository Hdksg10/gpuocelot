
#include <hip/hip_runtime.h>
// __device__ int my_LOP_0x54(int A, int B, int C){
//   int temp;
//   asm("lop3.b32 %0, %1, %2, %3, 0x54;" : "=r"(temp) : "r"(A), "r"(B), "r"(C));
//   return temp;
// }

// __global__ void testkernel(){

//   printf("A=true, B=false, C=true,   F=%d\n", my_LOP_0x54(true, false, true));
//   printf("A=true, B=false, C=false,  F=%d\n", my_LOP_0x54(true, false, false));
//   printf("A=false, B=false, C=false, F=%d\n", my_LOP_0x54(false, false, false));
// }

__global__ void test_lop3(unsigned* a, unsigned* b, unsigned* c, unsigned* d){
    asm("lop3.b32 %0, %1, %2, %3, 0xff;" : "=r"(*d) : "r"(*a), "r"(*b), "r"(*c));
}